



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> 
#include <time.h>

#define ROWS 227
#define COLS 227
#define UNDECLARE -999

//		d_outp[y*ROWS + x] = UNDECLARE;

__global__ void sobel_gpu(int *d_outp, int *d_inp, int *d_mask) 
{
  int x = threadIdx.x + blockIdx.x * blockDim.x; 
  int y = threadIdx.y + blockIdx.y * blockDim.y; 
  unsigned int idx = y*ROWS + x;
  
  if(idx < COLS * ROWS)
  {
	if( y!=0 || y!=(ROWS-1) || x!=0 || x!=(COLS-1))
	{	d_outp[y*ROWS + x] = d_inp[y*ROWS + x - 4] * d_mask[0] + 
							d_inp[y*ROWS + x - 3] * d_mask[1] +
							d_inp[y*ROWS + x - 2] * d_mask[2] +
							d_inp[y*ROWS + x - 1] * d_mask[3] +
							d_inp[y*ROWS + x    ] * d_mask[4] +
							d_inp[y*ROWS + x + 1] * d_mask[5] +
							d_inp[y*ROWS + x + 2] * d_mask[6] +
							d_inp[y*ROWS + x + 3] * d_mask[7] +
							d_inp[y*ROWS + x + 4] * d_mask[8];
	}
  }
}

void sobel_cpu(int *h_outp, int *h_inp, int *h_mask) 
{
	for(int i=1 ; i<ROWS-1 ; i++)
	{
		for(int j=1 ; j<COLS-1 ; j++)
		{
		    h_outp[i*ROWS + j] = h_inp[i*ROWS + j - 4] * h_mask[0]  + 
			                     h_inp[i*ROWS + j - 3] * h_mask[1]  +
				                 h_inp[i*ROWS + j - 2] * h_mask[2]  +
				                 h_inp[i*ROWS + j - 1] * h_mask[3]  +
				                 h_inp[i*ROWS + j    ] * h_mask[4]  +
				                 h_inp[i*ROWS + j + 1] * h_mask[5]  +
				                 h_inp[i*ROWS + j + 2] * h_mask[6]  +
				                 h_inp[i*ROWS + j + 3] * h_mask[7]  +
				                 h_inp[i*ROWS + j + 4] * h_mask[8] ;
		}
		
	}
}


int main(void)
{
	int *h_inp;
	int *h_outp;
	int *h_mask;
	
	int picSize = ROWS * COLS * sizeof(int);
	int maskSize = 3 * 3 * sizeof(int);
    
	h_inp = (int*)malloc(picSize);
    h_outp = (int*)malloc(picSize);
    h_mask = (int*)malloc(maskSize);	
	
	srand(time(NULL));
	for(int i=0 ; i<ROWS ; i++)
	{
		for(int j=0 ; j<COLS ; j++)
		{
		    h_inp[i*ROWS + j] = ((rand() % 10)+1);
		}
	}
	h_mask[0] = -1;
	h_mask[1] = 0;
	h_mask[2] = 1;
	h_mask[3] = -2;
	h_mask[4] = 0;
	h_mask[5] = 2;
	h_mask[6] = -1;
	h_mask[7] = 0;
	h_mask[8] = 1;
	
	
	int *d_inp;
    int *d_outp;
    int *d_mask;
	hipMalloc((void**)&d_inp, picSize);
	hipMalloc((void**)&d_outp, picSize);
	hipMalloc((void**)&d_mask, maskSize);
	
	hipMemcpy(d_inp, h_inp, picSize, hipMemcpyHostToDevice);
	hipMemcpy(d_outp, h_outp, picSize, hipMemcpyHostToDevice);
	hipMemcpy(d_mask, h_mask, maskSize, hipMemcpyHostToDevice);
	
	
	for(int i=0 ; i<ROWS ; i++)
	{
		for(int j=0 ; j<COLS ; j++)
		{
		    h_outp[i*ROWS + j] = 0;
		}
	}
	
  
	/*
	clock_t start_serial, end_serial;
	
	start_serial = clock();
	
	sobel_cpu(h_outp, h_inp, h_mask);
		
	end_serial = clock();
	
	printf("CPU(serial) time: %f s.\n",(end_serial-start_serial)/(float)CLOCKS_PER_SEC);
	
	for(int i=0 ; i<ROWS ; i++)
	{
		for(int j=0 ; j<COLS ; j++)
		{
			if( i==0 || i==(ROWS-1) || j==0 || j==(COLS-1))
			{
				h_outp[i*ROWS + j] = UNDECLARE;
			}
		}
	}
	*/	
		

	
	dim3 dimBlock(32, 32);
  dim3 dimGrid(  ((COLS+dimBlock.x-1)/dimBlock.x),  ((ROWS+dimBlock.y-1)/dimBlock.y)  );
 
 /* for part3 */
 // dim3 dimGrid(2,13);


	sobel_gpu<<<dimGrid, dimBlock>>>(d_outp, d_inp, d_mask);
	
	clock_t start_gpu, end_gpu;

	start_gpu = clock();

	hipMemcpy(h_outp,d_outp,picSize,hipMemcpyDeviceToHost);
	
    end_gpu = clock();	
	
	printf("GPU(parallel) time: %f s.\n",(end_gpu-start_gpu)/(float)CLOCKS_PER_SEC);
  printf("dimGrid.x %d dimBlock.x %d \n", dimGrid.x, dimBlock.x);	
  printf("dimGrid.y %d dimBlock.y %d \n", dimGrid.y, dimBlock.y);
  printf("dimGrid.z %d dimBlock.z %d \n", dimGrid.z, dimBlock.z);


	for(int i=0 ; i<ROWS ; i++)
	{
		for(int j=0 ; j<COLS ; j++)
		{
			if( i==0 || i==(ROWS-1) || j==0 || j==(COLS-1))
			{
				h_outp[i*ROWS + j] = UNDECLARE;
			}
		}
	}

	
	
	free( h_inp );
    free( h_outp );
    free( h_mask );
	
	hipFree(d_inp);
	hipFree(d_outp);
	hipFree(d_mask);
	
	return 0;
}